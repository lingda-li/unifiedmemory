#include "hip/hip_runtime.h"
#include <stdio.h>

#define DEVICE_ALLOC
//#define UVM_ALLOC
//#define HOST_ALLOC

//#define SIZE (2048 * 4)
//#define SIZE (1024 * 1024)
//#define SIZE (1024 * 1024 * 1024)
#define SIZE (1024 * 1024 * 1024L * 2)

__global__ void kernel(int *input)
{
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;

  input[i] += 1;
}

int main()
{
  int *d_input;

  hipEvent_t start;
  hipEvent_t end;

#if defined(DEVICE_ALLOC)
  hipMalloc(&d_input, SIZE*sizeof(int));
#elif defined(UVM_ALLOC)
  hipMallocManaged(&d_input, SIZE*sizeof(int));
#elif defined(HOST_ALLOC)
  hipHostMalloc(&d_input, SIZE*sizeof(int));
#else
  return 0;
#endif

  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start,0);

  kernel<<<1, 1>>>(d_input);

  hipDeviceSynchronize();
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, start, end);

  double AvgTP = (double)SIZE*sizeof(int)/(elapsed_time / 1000.0);
  printf("Average throughput: %f\n", AvgTP);

  hipFree(d_input);
  hipEventDestroy(start);
  hipEventDestroy(end);
  return 0;
}
