
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//#define DEVICE_ALLOC
#define UVM_ALLOC
//#define HOST_ALLOC

//#define SIZE (1024 * 8)
//#define STEP 16
//#define SIZE (1024 * 32)
//#define STEP 128

#define SIZE (1024 * 1024)
#define STEP (1024 * 32)

//#define SIZE (1024 * 1024 * 1024)
//#define STEP (1024 * 1024 * 32)
//#define STEP (512)

//#define SIZE (1024 * 1024 * 1024L * 5)
//#define SIZE (1024 * 1024 * 512L * 7)
//#define STEP (1024 * 1024 * 32)

#define PRINT_LAT

#define LAT_ARRAY_SIZE 12
#define LAT_LOWER_BOUND 3000
#define LAT_HIGHER_BOUND 10000

__global__ void warmup(int *input, int gid, int gnum)
{
  __shared__ int s_tmp;
  s_tmp = 0;

  for (unsigned long long i = (SIZE / gnum) * gid; i < (SIZE / gnum) * (gid + 1); i += STEP) {
    s_tmp += input[i];
  }
}

__global__ void kernel(int *input, double *total_lat, unsigned long long size)
{
  unsigned t0, t1, lat;
  __shared__ int s_tmp;
  int tmp;
  double maxlat, minlat, totallat;
  double maxlat_l, minlat_l, totallat_l;
  double maxlat_s, minlat_s, totallat_s;
  double llat_num, slat_num;

  s_tmp = 0;
  totallat = maxlat = minlat = 0.0;
  totallat_l = maxlat_l = minlat_l = 0.0;
  totallat_s = maxlat_s = minlat_s = 0.0;
  llat_num = slat_num = 0.0;

  for (unsigned long long i = 0; i < size; i += STEP) {
    t0 = clock();
    __syncthreads();
    tmp = input[i];
    __syncthreads();
    t1 = clock();
    lat = t1 - t0;
    s_tmp = tmp;
#ifdef PRINT_LAT
    printf("0x%10llx: %d\n", i, lat);
#endif
    totallat += lat;
    if (lat > maxlat)
      maxlat = lat;
    if (lat < minlat || minlat == 0)
      minlat = lat;

    // classify lat
    if (lat >= LAT_LOWER_BOUND && lat <= LAT_HIGHER_BOUND)
      total_lat[3] += lat;
    else if (lat < LAT_LOWER_BOUND) {
      totallat_s += lat;
      if (lat > maxlat_s)
        maxlat_s = lat;
      if (lat < minlat_s || minlat_s == 0)
        minlat_s = lat;
      slat_num++;
    } else {
      totallat_l += lat;
      if (lat > maxlat_l)
        maxlat_l = lat;
      if (lat < minlat_l || minlat_l == 0)
        minlat_l = lat;
      llat_num++;
    }
  }
  total_lat[0] = totallat;
  total_lat[1] = maxlat;
  total_lat[2] = minlat;

  total_lat[4] = totallat_l;
  total_lat[5] = maxlat_l;
  total_lat[6] = minlat_l;

  total_lat[7] = totallat_s;
  total_lat[8] = maxlat_s;
  total_lat[9] = minlat_s;

  total_lat[10] = llat_num;
  total_lat[11] = slat_num;
}

int main()
{
  int numGPUs = 0;
  hipGetDeviceCount(&numGPUs);
  printf("# GPUs: %d\n", numGPUs);
  int *d_input;
  double *total_lat, *h_total_lat;

  h_total_lat = (double*)malloc(LAT_ARRAY_SIZE * sizeof(double));
  hipMalloc(&total_lat, LAT_ARRAY_SIZE*sizeof(double));
  for (int i = 0; i < LAT_ARRAY_SIZE; i++)
    h_total_lat[i] = 0.0;
  hipMemcpy(total_lat, h_total_lat, LAT_ARRAY_SIZE*sizeof(double), hipMemcpyHostToDevice);

#if defined(DEVICE_ALLOC)
  cudaMalloc(&d_input, SIZE*sizeof(int));
#elif defined(UVM_ALLOC)
  hipMallocManaged(&d_input, SIZE*sizeof(int));
#elif defined(HOST_ALLOC)
  cudaMallocHost(&d_input, SIZE*sizeof(int));
#else
  return 0;
#endif

  // init
#if defined(DEVICE_ALLOC)
  int *h_input;
  h_input = (int*)malloc(SIZE*sizeof(int));
  for (unsigned long long i = 0; i < SIZE; i += STEP) {
    h_input[i] = rand();
  }
  cudaMemcpy(d_input, h_input, SIZE*sizeof(int), cudaMemcpyHostToDevice);
#elif defined(UVM_ALLOC) || defined(HOST_ALLOC)
  for (unsigned long long i = 0; i < SIZE; i += STEP) {
    d_input[i] = rand();
  }
#endif

  for (int d = 0; d < numGPUs; d++) {
    hipSetDevice(d);
    warmup<<<1, 1>>>(d_input, d, numGPUs);
    hipDeviceSynchronize();
  }

  hipSetDevice(0);
  kernel<<<1, 1>>>(d_input, total_lat, SIZE/2);
  hipDeviceSynchronize();

  hipMemcpy(h_total_lat, total_lat, LAT_ARRAY_SIZE*sizeof(double), hipMemcpyDeviceToHost);
  double AvgLat = h_total_lat[0] / (SIZE / STEP);
  printf("Average latency: %f (%f / %lld)\n", AvgLat, h_total_lat[0], SIZE / STEP);
  printf("Max latency: %f\n", h_total_lat[1]);
  printf("Min latency: %f\n", h_total_lat[2]);
  printf("\n");
  printf("Average latency (large): %f (%f / %f)\n", h_total_lat[4] / h_total_lat[10], h_total_lat[4], h_total_lat[10]);
  printf("Max latency (large): %f\n", h_total_lat[5]);
  printf("Min latency (large): %f\n", h_total_lat[6]);
  printf("\n");
  printf("Average latency (short): %f (%f / %f)\n", h_total_lat[7] / h_total_lat[11], h_total_lat[7], h_total_lat[11]);
  printf("Max latency (short): %f\n", h_total_lat[8]);
  printf("Min latency (short): %f\n", h_total_lat[9]);
  printf("\n");
  printf("Abnormal total: %f\n", h_total_lat[3]);

  kernel<<<1, 1>>>(&d_input[SIZE/2], total_lat, SIZE/2);
  hipDeviceSynchronize();

  hipMemcpy(h_total_lat, total_lat, LAT_ARRAY_SIZE*sizeof(double), hipMemcpyDeviceToHost);
  AvgLat = h_total_lat[0] / (SIZE / STEP);
  printf("Average latency: %f (%f / %lld)\n", AvgLat, h_total_lat[0], SIZE / STEP);
  printf("Max latency: %f\n", h_total_lat[1]);
  printf("Min latency: %f\n", h_total_lat[2]);
  printf("\n");
  printf("Average latency (large): %f (%f / %f)\n", h_total_lat[4] / h_total_lat[10], h_total_lat[4], h_total_lat[10]);
  printf("Max latency (large): %f\n", h_total_lat[5]);
  printf("Min latency (large): %f\n", h_total_lat[6]);
  printf("\n");
  printf("Average latency (short): %f (%f / %f)\n", h_total_lat[7] / h_total_lat[11], h_total_lat[7], h_total_lat[11]);
  printf("Max latency (short): %f\n", h_total_lat[8]);
  printf("Min latency (short): %f\n", h_total_lat[9]);
  printf("\n");
  printf("Abnormal total: %f\n", h_total_lat[3]);

  hipFree(d_input);
  hipFree(total_lat);
  return 0;
}
