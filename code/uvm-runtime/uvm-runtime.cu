#include "uvm-runtime.h"

//#define ALL_MANAGED

void uvmMalloc(struct uvmMallocInfo* uvmInfo)
{
  size_t size = uvmInfo->size;

#if defined (ALL_MANAGED)
  hipMallocManaged(&uvmInfo->devPtr, size);
  uvmInfo->hostPtr = uvmInfo->devPtr;
  uvmInfo->isSame = true;
#else
  uvmInfo->hostPtr = malloc(size);
  hipMalloc(&uvmInfo->devPtr, size);
  uvmInfo->isSame = false;
#endif
}

void uvmFree(struct uvmMallocInfo* uvmInfo)
{
  hipFree(uvmInfo->devPtr);
  if (!uvmInfo->isSame)
    free(uvmInfo->hostPtr);
}

void uvmMemcpy(struct uvmMallocInfo* uvmInfo, hipMemcpyKind kind)
{
  if (uvmInfo->isSame)
    return;
  void* devPtr = uvmInfo->devPtr;
  void* hostPtr = uvmInfo->hostPtr;
  size_t size = uvmInfo->size;

  if (kind == hipMemcpyHostToDevice)
    hipMemcpy(devPtr, hostPtr, size, kind);
  else
    hipMemcpy(hostPtr, devPtr, size, kind);
}
