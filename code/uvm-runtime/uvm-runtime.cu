#include "uvm-runtime.h"

#define ALL_MANAGED
#define GPU_PRE_PASCAL

void __uvm_malloc(struct uvmMallocInfo* uvmInfo)
{
  size_t size = uvmInfo->size;

#if defined (ALL_MANAGED)
  hipMallocManaged(&uvmInfo->devPtr, size);
  uvmInfo->hostPtr = uvmInfo->devPtr;
  uvmInfo->isSame = true;
#else
  uvmInfo->hostPtr = malloc(size);
  hipMalloc(&uvmInfo->devPtr, size);
  uvmInfo->isSame = false;
#endif
}

void __uvm_free(struct uvmMallocInfo* uvmInfo)
{
  hipFree(uvmInfo->devPtr);
  if (!uvmInfo->isSame)
    free(uvmInfo->hostPtr);
}

void __uvm_memcpy(struct uvmMallocInfo* uvmInfo, hipMemcpyKind kind)
{
  if (uvmInfo->isSame) {
#ifdef GPU_PRE_PASCAL
    if (kind == hipMemcpyDeviceToHost)
      hipDeviceSynchronize();
#endif
    return;
  }
  void* devPtr = uvmInfo->devPtr;
  void* hostPtr = uvmInfo->hostPtr;
  size_t size = uvmInfo->size;

  if (kind == hipMemcpyHostToDevice)
    hipMemcpy(devPtr, hostPtr, size, kind);
  else
    hipMemcpy(hostPtr, devPtr, size, kind);
}
