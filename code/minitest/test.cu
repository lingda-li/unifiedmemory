
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 10240

__global__ void MyKernel(int *a, int *b, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < size)
        b[id] = a[id] * 2;
}

int main()
{
    int i;
    int size = SIZE;
    int BlockSize = 256;
    int BlockNum = (size + BlockSize - 1) / BlockSize;
    int *a, *b;
    int *d_a, *d_b;

    a = (int*)malloc(size*sizeof(int));
    b = (int*)malloc(size*sizeof(int));
    for(i = 0; i < size; i++)
        a[i] = rand() % 100;

	hipMalloc((void **)&d_a, size*sizeof(int));
	hipMemcpy(d_a, a, size*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size*sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_b, size*sizeof(int));

    MyKernel<<<BlockNum, BlockSize>>>(d_a, d_b, size);

	hipMemcpy(b, d_b, size*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
